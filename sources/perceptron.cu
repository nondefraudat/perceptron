#include "hip/hip_runtime.h"
#include "perceptron.hxx"
#include <execution>
#include <sstream>
#include <fstream>
#include <cmath>

namespace neural_networks {

__global__ void activate(float data[], uint16_t size);

perceptron::perceptron(size_t layer_count) : _weights_count(layer_count - 1) {
	_biases = std::shared_ptr<gpu_math::vector[]>(
			new gpu_math::vector[_weights_count]);
	_weights = std::shared_ptr<gpu_math::matrix[]>(
			new gpu_math::matrix[_weights_count]);
}

perceptron::perceptron(std::initializer_list<uint16_t> layer_sizes)
		: perceptron(layer_sizes.size()) {
	if (layer_sizes.size() < 1) {
		throw std::invalid_argument("weights count must be more then 0");
	}
	const uint16_t *current_size = layer_sizes.begin();
	gpu_math::matrix *current_weights = _weights.get();
	gpu_math::vector *current_bias = _biases.get();
	uint16_t width = *current_size++;
	for (; current_size != layer_sizes.end();
			current_size++, current_weights++, current_bias++) {
		uint16_t height = *current_size;
		*current_weights = gpu_math::matrix(height, width, 1.f);
		*current_bias = gpu_math::vector(height, 0.f);
		width = height;
	}
}

perceptron perceptron::load(std::string dump_name) {
	std::stringstream ss;
	ss << std::ifstream(dump_name).rdbuf();
	size_t weights_count;
	ss >> weights_count;
	perceptron p(weights_count);
	for (size_t i = 0; i < weights_count; i++) {
		uint16_t height, width;
		ss >> height >> width;
		gpu_math::matrix &matrix = p._weights[i] =
				gpu_math::matrix(height, width);
		float value;
		for (uint16_t row = 0; row < height; row++) {
			for (uint16_t column = 0; column < width; column++) {
				ss >> value;
				matrix.set(row, column, value);
			}
		}
		gpu_math::vector &vector = p._biases[i] =
				gpu_math::vector(height);
		for (uint16_t index = 0; index < height; index++) {
			ss >> value;
			vector.set(index, value);
		}
	}
	return p;
}

void perceptron::dump(std::string dump_name) noexcept
{
	std::stringstream ss;
	ss << _weights_count << '\n';
	for (size_t i = 0; i < _weights_count; i++) {
		auto &weights = _weights[i];
		ss << weights.height() << ' ' << weights.width() << '\n';
		for (uint16_t row = 0; row < weights.height(); row++) {
			ss << weights.get(row, 0);
			for (uint16_t column = 1; column < weights.width(); column++) {
				ss << ' ' << weights.get(row, column);
			}
			ss << '\n';
		}
		auto &bias = _biases[i];
		ss << bias.get(0);
		for (uint16_t index = 1; index < weights.height(); index++) {
			ss << ' ' << bias.get(index);
		}
		ss << '\n';
	}
	std::ofstream(dump_name) << ss.str();
}

gpu_math::matrix perceptron::activate(const gpu_math::vector &signals) {
	gpu_math::matrix buffer, result = signals;
	for (size_t i = 0; i < _weights_count; i++) {
		buffer = _weights[i]*result + _biases[i];
		std::swap(buffer, result);
		neural_networks::activate<<<result.size(), 1>>>(
				result.device_data(), result.size());
	}
	return result;
}

__global__ void activate(float data[], uint16_t size) {
	auto index = blockIdx.x;
	if (index < size) {
		auto &value = data[index];
		value = 1/(1 + std::exp(-value));
	}
}

}